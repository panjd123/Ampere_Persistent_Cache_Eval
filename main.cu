/*
 * an example of persistent cache use case on A100
 * */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include <iostream>
#include <iomanip>

using namespace std;

const int block_num = 1024;
const int block_size = 1024;

#define ENABLE_PERSIST

__global__
void cuda_kernel(float *freq_data, float *stream_data, int freq_size, int stream_size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int num = (freq_size + stream_size) / block_size / block_num;
  for(int j = 0; j < num; j++) {
    freq_data[((i * num) + j) % freq_size] = freq_data[((i * num) + j) % freq_size] * 2;
    stream_data[((i * num) + j) % stream_size] = stream_data[((i * num) + j) % stream_size] * 2;
  }
}

int main(int argc, char** argv) {
  int device_id = 0;
  int data_size = 1024 * 1024 * 1024 / sizeof(float);
  int freq_size = 1024 * 1024 * 10 / sizeof(float);
  int stream_size = data_size - freq_size;
  int runs = 100;
  if(argc < 3){
    // printf("Usage: %s data_size(MB) freq_size(MB)\n", argv[0]);
    // exit(1);
    runs = 2;
  } else {
    data_size = atoi(argv[1]) * 1024 * 1024 / sizeof(float);
    freq_size = atoi(argv[2]) * 1024 * 1024 / sizeof(float);
    stream_size = data_size - freq_size;
    if(argc == 4) {
      runs = atoi(argv[3]);
    }
  }
  int warmup = 5;
  if(warmup >= runs) {
    warmup = runs - 1;
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  cudaCtxResetPersistingL2Cache();
  hipStream_t stream;
  hipStreamCreate(&stream);                                                                  // Create CUDA stream

  hipDeviceProp_t prop;                                                                        // CUDA device properties variable
  hipGetDeviceProperties(&prop, device_id);                                                 // Query GPU properties
  cout << "l2 cache size: " << prop.l2CacheSize << endl;
  cout << "max persisting cache size: " << prop.persistingL2CacheMaxSize << " Byte"<< endl;
  cout << "set persisting cache size: " << freq_size * sizeof(float) << " Byte"<< endl;
  hipDeviceSetLimit( cudaLimitPersistingL2CacheSize, min(freq_size * sizeof(float), static_cast<size_t>(prop.persistingL2CacheMaxSize)));   // set-aside 3/4 of L2 cache for persisting accesses or the max allowed

  float* h_data = (float *)malloc(data_size * sizeof(float));
  float* data;
  // init host data
  for (int i = 0; i < data_size; i++) {
    h_data[i] = 1.0f;
  }

  hipMalloc(&data, data_size * sizeof(float)); 
  hipMemcpy(data, h_data, data_size * sizeof(float), hipMemcpyHostToDevice);
  
 #ifdef ENABLE_PERSIST 
  hipLaunchAttributeValue stream_attribute;
  stream_attribute.accessPolicyWindow.base_ptr  = reinterpret_cast<void*>(data);              // Global Memory data pointer
  stream_attribute.accessPolicyWindow.num_bytes = min((long)(freq_size * sizeof(float)), (long)(prop.persistingL2CacheMaxSize));                  // Number of bytes for persistence access
  stream_attribute.accessPolicyWindow.hitRatio  = 1.0;                                        // Hint for cache hit ratio
  stream_attribute.accessPolicyWindow.hitProp   = hipAccessPropertyPersisting;               // Persistence Property
  stream_attribute.accessPolicyWindow.missProp  = hipAccessPropertyStreaming;                // Type of access property on cache miss
  
  cout << "window num_bytes: " << stream_attribute.accessPolicyWindow.num_bytes << endl;
  cout << "window hit ratio: " << stream_attribute.accessPolicyWindow.hitRatio << endl;
  
  hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);   // Set the attributes to a CUDA Stream
#endif
  {
    float accum = 0;
    for(int i = 0; i < runs; i++) {
      hipEventRecord(start);
      
      cuda_kernel <<<block_num, block_size, 0, stream>>> (data, data + freq_size, freq_size, stream_size); // This data1 is used by a kernel multiple times
      hipEventRecord(stop);
      // copy results
      hipMemcpy(h_data, data, data_size * sizeof(float), hipMemcpyDeviceToHost);
      hipEventSynchronize(stop);
      float milliseconds = 0;
      hipEventElapsedTime(&milliseconds, start, stop);
      accum += milliseconds;
      if(i >= warmup){
        accum += milliseconds;
      }
    }
    cout << "Time: " << fixed << setprecision(6) << accum / (runs - warmup) << " ms" << endl;
    hipStreamDestroy(stream);
    cudaCtxResetPersistingL2Cache();
  }
  {
    float accum = 0;
    for(int i = 0; i < runs; i++) {
      hipEventRecord(start);
      
      cuda_kernel <<<block_num, block_size>>> (data, data + freq_size, freq_size, stream_size); // This data1 is used by a kernel multiple times
      hipEventRecord(stop);
      // copy results
      hipMemcpy(h_data, data, data_size * sizeof(float), hipMemcpyDeviceToHost);
      hipEventSynchronize(stop);
      float milliseconds = 0;
      hipEventElapsedTime(&milliseconds, start, stop);
      accum += milliseconds;
      if(i >= warmup){
        accum += milliseconds;
      }
    }
    cout << "Time: " << fixed << setprecision(6) << accum / (runs - warmup) << " ms" << endl;
  }
}


